#include "CTheoMat.cuh"
#include <iostream>
#include <random>
#include <hipblas.h>


void Theo::CTheoMat::hello() {
    std::cout << "Hello I am the theo's custom matrix library, WIP" << std::endl;
}

double* Theo::CTheoMat::initMat() const {
    auto matTemp = new double[rows * columns];
    for (int i = 0; i < rows * columns ; i++){
            matTemp[i] = 0;
    }
    return matTemp;
}

Theo::CTheoMat::CTheoMat(): rows(0), columns(0), mat(initMat()) {}

Theo::CTheoMat::CTheoMat(int rows_, int columns_): rows(rows_), columns(columns_), mat(initMat()) {}

Theo::CTheoMat::CTheoMat(const Theo::CTheoMat& matrix): rows(matrix.getN()), columns(matrix.getM()), mat(initMat()) {
    // deep copy, maybe not the best way to do it
    for(int i = 0; i < rows * columns; i++){
        mat[i] = matrix[i];
    }
}

Theo::CTheoMat::CTheoMat(std::initializer_list<std::initializer_list<double>> initList) {
    rows = initList.size();
    columns = initList.begin()->size();
    mat = initMat();
    int i = 0, j = 0;
    for(auto & row : initList) {
        if(row.size() != columns) {
            throw std::out_of_range("invalid initialisation list size");
        }
        for(auto & value : row){
            mat[i * columns + j] = value;
            j++;
        }
        j = 0;
        i++;
    }
}

Theo::CTheoMat::CTheoMat(std::vector<std::vector<double>> initVec) {
    rows = initVec.size();
    columns = initVec.begin()->size();
    mat = initMat();
    int i = 0, j = 0;
    for(auto & row : initVec) {
        if(row.size() != columns) {
            throw std::out_of_range("invalid initialisation vector size");
        }
        for(auto & value : row){
            mat[i * columns + j] = value;
            j++;
        }
        j = 0;
        i++;
    }
}

/**
 * Creates a one line matrix with the given vector
 * @param initVect
 */
Theo::CTheoMat::CTheoMat(std::vector<double> initVect) {
    rows = 1;
    columns = initVect.size();
    mat = initMat();
    int i = 0;
    for(auto & value : initVect){
        mat[i] = value;
        i++;
    }
}

/**
 * Creates a matrix with just one line using the given values
 * @param values
 * @param size
 */
Theo::CTheoMat::CTheoMat(double *values, int size): rows(1), columns(size), mat(initMat()){
    for(int i = 0; i < columns; i++){
        mat[i] = values[i];
    }
}


void Theo::CTheoMat::freeMat() {
    delete [] mat;
}

Theo::CTheoMat::~CTheoMat() {
    freeMat();
}

double Theo::CTheoMat::getValue(int i, int j) const {
    if(i >= 0 && i < rows && j >= 0 && j < columns) {
        return (double) mat[i * columns + j];
    }
    throw std::out_of_range("i and/or j out of matrix range");
}

void Theo::CTheoMat::setValue(double value, int i, int j) {
    if(i >= 0 && i < rows && j >= 0 && j < columns){
        mat[i * columns + j] = value;
    }
}

void Theo::CTheoMat::random() {
    std::random_device r;
    std::default_random_engine e1(r());
    std::uniform_real_distribution<double> uniformDist(0,1);
    for(int i = 0; i < rows * columns; i++){
        mat[i] = uniformDist(e1);
    }
}

int Theo::CTheoMat::getN() const {return rows;}

int Theo::CTheoMat::getM() const {return columns;}

std::string Theo::CTheoMat::toString() {
    std::string s = "[";
    for(int i = 0; i < rows; i++){
        s += "[";
        for(int j = 0; j < columns - 1; j++){
            s += std::to_string(mat[i * columns + j]); s += ", ";
        }
        s += std::to_string(mat[i * columns + columns - 1]);
        s += "]";
        if(i != rows - 1){ s += "\n";}
    }
    s += "]";
    return s;
}

bool Theo::CTheoMat::checkDim(const Theo::CTheoMat &matrix) const {
    return rows == matrix.getN() && columns == matrix.getM();
}

Theo::CTheoMat & Theo::CTheoMat::operator=(const Theo::CTheoMat &matrix) {
    if(this != &matrix){
        freeMat();
        rows = matrix.getN();
        columns = matrix.getM();
        mat = new double[rows * columns];
        for (int i = 0; i < rows * columns; i++){
            mat[i] = matrix[i];
        }
    }
    return *this;
}

Theo::CTheoMat Theo::CTheoMat::operator+(const Theo::CTheoMat& matrix) {
    if(checkDim(matrix)){
        CTheoMat result(rows, columns);
        for(int i = 0; i < rows * columns; i++){
            result[i] = mat[i] + matrix[i];
        }
        return result;
    }
    std::string errorMessage = "cannot add matrices of different sizes (" + std::to_string(rows) + ", "
                               + std::to_string(columns) + ") and (" + std::to_string(matrix.getN()) + ", "
                               + std::to_string(matrix.getM()) + ")\n";
    throw std::out_of_range(errorMessage);
}

Theo::CTheoMat & Theo::CTheoMat::operator+=(const Theo::CTheoMat &matrix) {
    *this = *this + matrix;
    return *this;
}

Theo::CTheoMat Theo::CTheoMat::operator-(const Theo::CTheoMat& matrix) {
    if(checkDim(matrix)) {
        CTheoMat result(rows, columns);
        for (int i = 0; i < rows * columns; i++) {
            result[i] = mat[i] - matrix[i];
        }
        return result;
    }
    std::string errorMessage = "cannot subtract matrices of different sizes (" + std::to_string(rows) + ", "
                               + std::to_string(columns) + ") and (" + std::to_string(matrix.getN()) + ", "
                               + std::to_string(matrix.getM()) + ")\n";
    throw std::out_of_range(errorMessage);
}

Theo::CTheoMat Theo::CTheoMat::operator*(const Theo::CTheoMat &matrix) const {
    // very slow implementation ...
    if(columns == matrix.getN()) {
        CTheoMat result(rows, matrix.getM());
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < matrix.getM(); j++) {
                auto s = 0;
                for(int k = 0; k < columns; k++){
                    s += mat[i * columns + k] * matrix(k, j);
                }
                result(i, j) = s;
            }
        }
        return result;
    }
    std::string errorMessage = "cannot multiply incompatible matrices. first one has " + std::to_string(columns) +
                               " columns while second one has " + std::to_string(matrix.getN()) + " rows \n";
    throw std::out_of_range(errorMessage);

}

Theo::CTheoMat Theo::CTheoMat::operator*(double k) const {
    CTheoMat result(rows, columns);
    for(int i = 0; i < rows * columns; i++){
            result[i] = mat[i] * k;
    }
    return result;
}

Theo::CTheoMat Theo::CTheoMat::operator/(double k) const {
    CTheoMat result(rows, columns);
    for(int i = 0; i < rows * columns; i++){
        result[i] = mat[i] / k;
    }
    return result;
}

double& Theo::CTheoMat::operator[](int i) {
    if(i >= rows * columns){
        throw std::out_of_range("out of matrix range");
    }
    return mat[i];
}

double& Theo::CTheoMat::operator[](int& i) const {
    if(i >= rows * columns){
        throw std::out_of_range("out of matrix range");
    }
    return mat[i];
}

double &Theo::CTheoMat::operator()(int i, int j) {
    if(i >= 0 && i < rows && j >= 0 && j < columns) {
        return mat[i * columns + j];
    }
    throw std::out_of_range("i or j is out of the matrix range");
}

double &Theo::CTheoMat::operator()(int &i, int &j) const {
    if(i >= 0 && i < rows && j >= 0 && j < columns) {
        return mat[i * columns + j];
    }
    throw std::out_of_range("i or j is out of the matrix range");
}

bool Theo::CTheoMat::operator==(const Theo::CTheoMat &matrix) const {
    if(rows != matrix.getN() || columns != matrix.getM()){
        return false;
    }
    for(int i = 0; i < rows * columns; i++){
        if(mat[i] != matrix[i]){
            return false;
        }
    }
    return true;
}

Theo::CTheoMat Theo::CTheoMat::transpose() {
    CTheoMat transpose(columns, rows);
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < columns; j++){
            transpose(j, i) = mat[i * columns + j];
        }
    }
    return transpose;
}

Theo::CTheoMat Theo::CTheoMat::identity(int size) {
    CTheoMat id(size, size);
    for(int i = 0; i < size; i++){
        id(i, i) = 1;
    }
    return id;
}

Theo::CTheoMat Theo::operator*(const double k, const Theo::CTheoMat &matrix) {
    return matrix * k;
}
