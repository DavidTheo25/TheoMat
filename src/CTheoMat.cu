#include "CTheoMat.cuh"
#include <iostream>
#include <random>
#include <hipblas.h>


void Theo::CTheoMat::hello() {
    std::cout << "Hello I am the theo's custom matrix library, WIP" << std::endl;
}

float* Theo::CTheoMat::initMat() const {
    auto matTemp = new float[rows * columns];
//    float* matTemp;
//    hipMallocManaged(&matTemp, rows * columns * sizeof(float));
    for (int i = 0; i < rows * columns ; i++){
            matTemp[i] = 0;
    }
    return matTemp;
}

Theo::CTheoMat::CTheoMat(): rows(0), columns(0), mat(initMat()) {}

Theo::CTheoMat::CTheoMat(int rows_, int columns_): rows(rows_), columns(columns_), mat(initMat()) {}

Theo::CTheoMat::CTheoMat(const Theo::CTheoMat& matrix): rows(matrix.getRows()), columns(matrix.getColumns()), mat(initMat()) {
    // deep copy, maybe not the best way to do it
    for(int i = 0; i < rows * columns; i++){
        mat[i] = matrix[i];
    }
}

Theo::CTheoMat::CTheoMat(std::initializer_list<std::initializer_list<float>> initList) {
    columns = initList.size();
    rows = initList.begin()->size();
    mat = initMat();
    int i = 0, j = 0;
    for(auto & column : initList) {
        if(column.size() != rows) {
            throw std::out_of_range("invalid initialisation list size");
        }
        for(auto & value : column){
            mat[i * rows + j] = value;
            j++;
        }
        j = 0;
        i++;
    }
}

Theo::CTheoMat::CTheoMat(std::vector<std::vector<float>> initVec) {
    columns = initVec.size();
    rows = initVec.begin()->size();
    mat = initMat();
    int i = 0, j = 0;
    for(auto & column : initVec) {
        if(column.size() != rows) {
            throw std::out_of_range("invalid initialisation list size");
        }
        for(auto & value : column){
            mat[i * rows + j] = value;
            j++;
        }
        j = 0;
        i++;
    }
}

/**
 * Creates a one line matrix with the given vector
 * @param initVect
 */
Theo::CTheoMat::CTheoMat(std::vector<float> initVect) {
    columns = 1;
    rows = initVect.size();
    mat = initMat();
    int i = 0;
    for(auto & value : initVect){
        mat[i] = value;
        i++;
    }
}

/**
 * Creates a matrix with just one line using the given values
 * @param values
 * @param size
 */
Theo::CTheoMat::CTheoMat(float *values, int size): rows(size), columns(1), mat(initMat()){
    for(int i = 0; i < columns; i++){
        mat[i] = values[i];
    }
}


void Theo::CTheoMat::freeMat() {
    delete [] mat;
//    hipFree(mat);
}

Theo::CTheoMat::~CTheoMat() {
    freeMat();
}

float Theo::CTheoMat::getValue(int i, int j) const {
    if(i >= 0 && i < columns && j >= 0 && j < rows) {
        return (float) mat[i * rows + j];
    }
    throw std::out_of_range("i and/or j out of matrix range");
}

void Theo::CTheoMat::setValue(float value, int i, int j) {
    if(i >= 0 && i < columns && j >= 0 && j < rows){
        mat[i * rows + j] = value;
    }
}

void Theo::CTheoMat::random() {
    //this could be done with hiprand
    std::random_device r;
    std::default_random_engine e1(r());
    std::uniform_real_distribution<float> uniformDist(0,1);
    for(int i = 0; i < rows * columns; i++){
        mat[i] = uniformDist(e1);
    }
}

int Theo::CTheoMat::getRows() const {return rows;}

int Theo::CTheoMat::getColumns() const {return columns;}

std::string Theo::CTheoMat::toString() {
    std::string s = "[";
    for (int j = 0; j < rows; j++){
        s += "[";
        for (int i = 0; i < columns - 1; i++){
            s += std::to_string(mat[i * rows + j]) + ", ";
        }
        s += std::to_string(mat[(columns - 1) * rows + j]);
        s += "]\n";
    }
    s.pop_back();
    s += "]\n";
    return s;
}

bool Theo::CTheoMat::checkDim(const Theo::CTheoMat &matrix) const {
    return rows == matrix.getRows() && columns == matrix.getColumns();
}

Theo::CTheoMat & Theo::CTheoMat::operator=(const Theo::CTheoMat &matrix) {
    if(this != &matrix){
        freeMat();
        rows = matrix.getRows();
        columns = matrix.getColumns();
        mat = new float[rows * columns];
//        hipMallocManaged(&mat, rows * columns * sizeof(float));
        for (int i = 0; i < rows * columns; i++){
            mat[i] = matrix[i];
        }
    }
    return *this;
}

Theo::CTheoMat Theo::CTheoMat::operator+(const Theo::CTheoMat& matrix) {
    if(checkDim(matrix)){
        CTheoMat result(rows, columns);
        for(int i = 0; i < rows * columns; i++){
            result[i] = mat[i] + matrix[i];
        }
        return result;
    }
    std::string errorMessage = "cannot add matrices of different sizes (" + std::to_string(rows) + ", "
                               + std::to_string(columns) + ") and (" + std::to_string(matrix.getRows()) + ", "
                               + std::to_string(matrix.getColumns()) + ")\n";
    throw std::out_of_range(errorMessage);
}

Theo::CTheoMat & Theo::CTheoMat::operator+=(const Theo::CTheoMat &matrix) {
    *this = *this + matrix;
    return *this;
}

Theo::CTheoMat Theo::CTheoMat::operator-(const Theo::CTheoMat& matrix) {
    if(checkDim(matrix)) {
        CTheoMat result(rows, columns);
        for (int i = 0; i < rows * columns; i++) {
            result[i] = mat[i] - matrix[i];
        }
        return result;
    }
    std::string errorMessage = "cannot subtract matrices of different sizes (" + std::to_string(rows) + ", "
                               + std::to_string(columns) + ") and (" + std::to_string(matrix.getRows()) + ", "
                               + std::to_string(matrix.getColumns()) + ")\n";
    throw std::out_of_range(errorMessage);
}

Theo::CTheoMat Theo::CTheoMat::operator*(const Theo::CTheoMat &matrix) const {
    // very slow implementation ...
    int M = rows;
    int N = matrix.getColumns();
    int K = columns;
    if(columns == matrix.getRows()) {
        CTheoMat result(M, N);

        // Pre-calculate the size (in bytes) of our matrices
        const size_t bytes_a = M * K * sizeof(float);
        const size_t bytes_b = K * N * sizeof(float);
        const size_t bytes_c = M * N * sizeof(float);

        // Allocate device memory
        float *d_a, *d_b, *d_c;
        hipMalloc(&d_a, bytes_a);
        hipMalloc(&d_b, bytes_b);
        hipMalloc(&d_c, bytes_c);

        hipMemcpy(d_a, mat, bytes_a, hipMemcpyHostToDevice);
        hipMemcpy(d_b, matrix.mat, bytes_b, hipMemcpyHostToDevice);
        hipMemcpy(d_c, result.mat, bytes_c, hipMemcpyHostToDevice);

        // cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // Scalaing factors
        float alpha = 1.0f;
        float beta = 0.0f;

        // Calculate: c = (alpha*a) * b + (beta*c)
        // MxN = MxK * KxN
        // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
        // beta, C, ldc
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, K,
                    &beta, d_c, M);

        hipMemcpy(result.mat, d_c, bytes_c, hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return result;

    }
    std::string errorMessage = "cannot multiply incompatible matrices. first one has " + std::to_string(columns) +
                               " columns while second one has " + std::to_string(matrix.getRows()) + " rows \n";
    throw std::out_of_range(errorMessage);

}

Theo::CTheoMat Theo::CTheoMat::operator*(float k) const {
    CTheoMat result(rows, columns);
    for(int i = 0; i < rows * columns; i++){
            result[i] = mat[i] * k;
    }
    return result;
}

Theo::CTheoMat Theo::CTheoMat::operator/(float k) const {
    CTheoMat result(rows, columns);
    for(int i = 0; i < rows * columns; i++){
        result[i] = mat[i] / k;
    }
    return result;
}

float& Theo::CTheoMat::operator[](int i) {
    if(i >= rows * columns){
        throw std::out_of_range("out of matrix range");
    }
    return mat[i];
}

float& Theo::CTheoMat::operator[](int& i) const {
    if(i >= rows * columns){
        throw std::out_of_range("out of matrix range");
    }
    return mat[i];
}

float &Theo::CTheoMat::operator()(int i, int j) {
    if(i >= 0 && i < columns && j >= 0 && j < rows) {
        return mat[i * rows + j];
    }
    throw std::out_of_range("i or j is out of the matrix range");
}

float &Theo::CTheoMat::operator()(int &i, int &j) const {
    if(i >= 0 && i < columns && j >= 0 && j < rows) {
        return mat[i * rows + j];
    }
    throw std::out_of_range("i or j is out of the matrix range");
}

bool Theo::CTheoMat::operator==(const Theo::CTheoMat &matrix) const {
    if(rows != matrix.getRows() || columns != matrix.getColumns()){
        return false;
    }
    for(int i = 0; i < rows * columns; i++){
        if(mat[i] != matrix[i]){
            return false;
        }
    }
    return true;
}

Theo::CTheoMat Theo::CTheoMat::transpose() {
    CTheoMat transpose(columns, rows);
    for(int i = 0; i < columns; i++){
        for(int j = 0; j < rows; j++){
            transpose(j, i) = mat[i * rows + j];
        }
    }
    return transpose;
}

Theo::CTheoMat Theo::CTheoMat::identity(int size) {
    CTheoMat id(size, size);
    for(int i = 0; i < size; i++){
        id(i, i) = 1;
    }
    return id;
}

Theo::CTheoMat Theo::operator*(const float k, const Theo::CTheoMat &matrix) {
    return matrix * k;
}
