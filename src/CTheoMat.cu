#include "CTheoMat.cuh"
#include <iostream>
#include <random>
#include <hipblas.h>


void Theo::CTheoMat::hello() {
    std::cout << "Hello I am the theo's custom matrix library, WIP" << std::endl;
}

double* Theo::CTheoMat::initMat() const {
    auto matTemp = new double[rows * columns];
    for (int i = 0; i < rows * columns ; i++){
            matTemp[i] = 0;
    }
    return matTemp;
}

Theo::CTheoMat::CTheoMat(): rows(0), columns(0), mat(initMat()) {}

Theo::CTheoMat::CTheoMat(int rows_, int columns_): rows(rows_), columns(columns_), mat(initMat()) {}

Theo::CTheoMat::CTheoMat(const Theo::CTheoMat& matrix): rows(matrix.getRows()), columns(matrix.getColumns()), mat(initMat()) {
    // deep copy, maybe not the best way to do it
    for(int i = 0; i < rows * columns; i++){
        mat[i] = matrix[i];
    }
}

Theo::CTheoMat::CTheoMat(std::initializer_list<std::initializer_list<double>> initList) {
    columns = initList.size();
    rows = initList.begin()->size();
    mat = initMat();
    int i = 0, j = 0;
    for(auto & column : initList) {
        if(column.size() != rows) {
            throw std::out_of_range("invalid initialisation list size");
        }
        for(auto & value : column){
            mat[i * rows + j] = value;
            j++;
        }
        j = 0;
        i++;
    }
}

Theo::CTheoMat::CTheoMat(std::vector<std::vector<double>> initVec) {
    columns = initVec.size();
    rows = initVec.begin()->size();
    mat = initMat();
    int i = 0, j = 0;
    for(auto & column : initVec) {
        if(column.size() != rows) {
            throw std::out_of_range("invalid initialisation list size");
        }
        for(auto & value : column){
            mat[i * rows + j] = value;
            j++;
        }
        j = 0;
        i++;
    }
}

/**
 * Creates a one line matrix with the given vector
 * @param initVect
 */
Theo::CTheoMat::CTheoMat(std::vector<double> initVect) {
    columns = 1;
    rows = initVect.size();
    mat = initMat();
    int i = 0;
    for(auto & value : initVect){
        mat[i] = value;
        i++;
    }
}

/**
 * Creates a matrix with just one line using the given values
 * @param values
 * @param size
 */
Theo::CTheoMat::CTheoMat(double *values, int size): rows(size), columns(1), mat(initMat()){
    for(int i = 0; i < columns; i++){
        mat[i] = values[i];
    }
}


void Theo::CTheoMat::freeMat() {
    delete [] mat;
}

Theo::CTheoMat::~CTheoMat() {
    freeMat();
}

double Theo::CTheoMat::getValue(int i, int j) const {
    if(i >= 0 && i < columns && j >= 0 && j < rows) {
        return (double) mat[i * rows + j];
    }
    throw std::out_of_range("i and/or j out of matrix range");
}

void Theo::CTheoMat::setValue(double value, int i, int j) {
    if(i >= 0 && i < columns && j >= 0 && j < rows){
        mat[i * rows + j] = value;
    }
}

void Theo::CTheoMat::random() {
    //this could be done with hiprand
    std::random_device r;
    std::default_random_engine e1(r());
    std::uniform_real_distribution<double> uniformDist(0,1);
    for(int i = 0; i < rows * columns; i++){
        mat[i] = uniformDist(e1);
    }
}

int Theo::CTheoMat::getRows() const {return rows;}

int Theo::CTheoMat::getColumns() const {return columns;}

std::string Theo::CTheoMat::toString() {
    std::string s = "[";
    for (int j = 0; j < rows; j++){
        s += "[";
        for (int i = 0; i < columns - 1; i++){
            s += std::to_string(mat[i * rows + j]) + ", ";
        }
        s += std::to_string(mat[(columns - 1) * rows + j]);
        s += "]\n";
    }
    s.pop_back();
    s += "]\n";
    return s;
}

bool Theo::CTheoMat::checkDim(const Theo::CTheoMat &matrix) const {
    return rows == matrix.getRows() && columns == matrix.getColumns();
}

Theo::CTheoMat & Theo::CTheoMat::operator=(const Theo::CTheoMat &matrix) {
    if(this != &matrix){
        freeMat();
        rows = matrix.getRows();
        columns = matrix.getColumns();
        mat = new double[rows * columns];
        for (int i = 0; i < rows * columns; i++){
            mat[i] = matrix[i];
        }
    }
    return *this;
}

Theo::CTheoMat Theo::CTheoMat::operator+(const Theo::CTheoMat& matrix) {
    if(checkDim(matrix)){
        CTheoMat result(rows, columns);
        for(int i = 0; i < rows * columns; i++){
            result[i] = mat[i] + matrix[i];
        }
        return result;
    }
    std::string errorMessage = "cannot add matrices of different sizes (" + std::to_string(rows) + ", "
                               + std::to_string(columns) + ") and (" + std::to_string(matrix.getRows()) + ", "
                               + std::to_string(matrix.getColumns()) + ")\n";
    throw std::out_of_range(errorMessage);
}

Theo::CTheoMat & Theo::CTheoMat::operator+=(const Theo::CTheoMat &matrix) {
    *this = *this + matrix;
    return *this;
}

Theo::CTheoMat Theo::CTheoMat::operator-(const Theo::CTheoMat& matrix) {
    if(checkDim(matrix)) {
        CTheoMat result(rows, columns);
        for (int i = 0; i < rows * columns; i++) {
            result[i] = mat[i] - matrix[i];
        }
        return result;
    }
    std::string errorMessage = "cannot subtract matrices of different sizes (" + std::to_string(rows) + ", "
                               + std::to_string(columns) + ") and (" + std::to_string(matrix.getRows()) + ", "
                               + std::to_string(matrix.getColumns()) + ")\n";
    throw std::out_of_range(errorMessage);
}

Theo::CTheoMat Theo::CTheoMat::operator*(const Theo::CTheoMat &matrix) const {
    // very slow implementation ...
    int M = rows;
    int N = matrix.getColumns();
    int K = columns;
    if(columns == matrix.getRows()) {
        CTheoMat result(M, N);
        for (int row = 0; row < M; row++) {
            for (int col = 0; col < N; col++) {
                auto s = 0;
                for(int i = 0; i < K; i++){
                    s += mat[row * K + i] * matrix(i, col);
                }
                result(row, col) = s;
            }
        }
        return result;
    }
    std::string errorMessage = "cannot multiply incompatible matrices. first one has " + std::to_string(columns) +
                               " columns while second one has " + std::to_string(matrix.getRows()) + " rows \n";
    throw std::out_of_range(errorMessage);

}

Theo::CTheoMat Theo::CTheoMat::operator*(double k) const {
    CTheoMat result(rows, columns);
    for(int i = 0; i < rows * columns; i++){
            result[i] = mat[i] * k;
    }
    return result;
}

Theo::CTheoMat Theo::CTheoMat::operator/(double k) const {
    CTheoMat result(rows, columns);
    for(int i = 0; i < rows * columns; i++){
        result[i] = mat[i] / k;
    }
    return result;
}

double& Theo::CTheoMat::operator[](int i) {
    if(i >= rows * columns){
        throw std::out_of_range("out of matrix range");
    }
    return mat[i];
}

double& Theo::CTheoMat::operator[](int& i) const {
    if(i >= rows * columns){
        throw std::out_of_range("out of matrix range");
    }
    return mat[i];
}

double &Theo::CTheoMat::operator()(int i, int j) {
    if(i >= 0 && i < columns && j >= 0 && j < rows) {
        return mat[i * rows + j];
    }
    throw std::out_of_range("i or j is out of the matrix range");
}

double &Theo::CTheoMat::operator()(int &i, int &j) const {
    if(i >= 0 && i < columns && j >= 0 && j < rows) {
        return mat[i * rows + j];
    }
    throw std::out_of_range("i or j is out of the matrix range");
}

bool Theo::CTheoMat::operator==(const Theo::CTheoMat &matrix) const {
    if(rows != matrix.getRows() || columns != matrix.getColumns()){
        return false;
    }
    for(int i = 0; i < rows * columns; i++){
        if(mat[i] != matrix[i]){
            return false;
        }
    }
    return true;
}

Theo::CTheoMat Theo::CTheoMat::transpose() {
    CTheoMat transpose(columns, rows);
    for(int i = 0; i < columns; i++){
        for(int j = 0; j < rows; j++){
            transpose(j, i) = mat[i * rows + j];
        }
    }
    return transpose;
}

Theo::CTheoMat Theo::CTheoMat::identity(int size) {
    CTheoMat id(size, size);
    for(int i = 0; i < size; i++){
        id(i, i) = 1;
    }
    return id;
}

Theo::CTheoMat Theo::operator*(const double k, const Theo::CTheoMat &matrix) {
    return matrix * k;
}
